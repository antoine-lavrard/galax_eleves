
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU
#include <iostream>
#include "cuda.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)
#include <stdio.h>
//require computing capability 5.0 or higher

__device__ float3 operator-(const float3 &a, const float3 &b) {

  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);

}

__device__ float3 operator+(const float3 &a, const float3 &b) {

  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);

}



__device__ float3 operator*(const float3 &a, const float3 &b) {

  return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);

}

__device__ float3 operator*(const float3 &a, const float b) {

  return make_float3(a.x*b, a.y*b, a.z*b);

}

__device__ float sum(const float3 &a) {

  return a.x + a.y + a.z;
}


__device__ void compute_difference(float3 &posi, float3 &posj, float3 &diff, float &dij)
{
	diff= posj - posi;
	dij=sum(diff*diff);
}




__device__ void compute_forces(const float &mi,const float &mj,float &dij, float &dij_mi,float &dij_mj){
    if (dij > 1)
    {
		dij=rsqrt(dij*dij*dij);
        dij_mj = dij * mj;
        dij_mi = dij * mi;
    }
    else
    {
        dij_mj = mj;
        dij_mi = mi;

    }
}

__global__ void update_acc(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU,float* massesGPU,const int n_particles)
{	
	//const int number_threads=blockDim.x;
	//int thread_id=threadIdx.x;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (i >= n_particles) return;
	float3 diff;
	float dij;
	float dij_mi;
	float dij_mj;
	printf("ok");
	
	for(int j =0; j<i; j++){ // note that j<i, ineficient on gpu
		
		compute_difference(positionsGPU[i], positionsGPU[j], diff, dij);
		compute_forces(massesGPU[i], massesGPU[j], dij, dij_mi, dij_mj);
		float3 to_add=diff*dij_mi;
		// TODO : aggregate accelerations between threads
		
	}


	printf(" acceleration : %f",accelerationsGPU[0].x);

}


// __global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
// {
// 	// unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

// }

static inline int divup(int a, int b) {
	// how manyœblocks of size b should we use to represent a block of size a
	return (a + b - 1)/b;
}



void update_position_cu(float3* positionsGPU,float3* velocitiesGPU, float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	// n blocks on the diagonal
	// n(n-1)/2 blocks on the lower triangle
	// launch diagonal blocks first,
	// then launch full blocks

	int nthreads = 128;
	int nblocks = divup(n_particles, nthreads);
	std::cout << "nblocks: " << nblocks << std::endl;
	std::cout << "nthreads: " << nthreads << std::endl;
	std::cout << "n_particles: " << n_particles << std::endl;

	update_acc <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU,\
	 massesGPU, n_particles);

	

}


#endif // GALAX_MODEL_GPU