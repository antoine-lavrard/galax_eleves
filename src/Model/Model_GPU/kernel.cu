
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU
#include <iostream>
#include "cuda.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)

__device__ float3 operator-(const float3 &a, const float3 &b) {

  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);

}

__device__ float3 operator+(const float3 &a, const float3 &b) {

  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);

}



__device__ float3 operator*(const float3 &a, const float3 &b) {

  return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);

}

__device__ float3 operator*(const float3 &a, const float b) {

  return make_float3(a.x*b, a.y*b, a.z*b);

}

__device__ float sum(const float3 &a) {

  return a.x + a.y + a.z;
}


__device__ void compute_difference(float3 &posi, float3 &posj, float3 &diff, float &dij)
{
	diff= posj - posi;
	dij=sum(diff*diff);
}




__device__ void compute_forces(const float &mi,const float &mj,float &dij, float &dij_mi,float &dij_mj){
    if (dij > 1)
    {
		dij=rsqrt(dij*dij*dij);
		dij_mi = dij * mj;
        dij_mj = dij * mi;
        
    }
    else
    {
		dij_mi = mj;
        dij_mj = mi;
    }
}

__device__ void compute_forces(const float &mj,float &dij, float &dij_mj){
    if (dij > 1)
    {
		dij=rsqrt(dij*dij*dij);
        dij_mj = dij * mj;
    }
    else
    {
        dij_mj = mj;
    }
}

template<const int number_unrolling>
__device__ void update_acc_at_i_between_j(float3 &position_at_i,float3 &accelerationsGPU_at_i,\
	float3 * positionsGPU,float* massesGPU,\
	 const int min_j, const int max_j){
	
	for(int j =min_j; j+number_unrolling<max_j; j+=number_unrolling){ 

		float3 diff[number_unrolling];
		float dij[number_unrolling];
		float dij_mj[number_unrolling];
		// can be unrolled since number_unrolling is a constant
		for (int k=0; k<number_unrolling; k++){
			compute_difference(position_at_i, positionsGPU[j+k], diff[k], dij[k]);
			compute_forces(massesGPU[j+k], dij[k], dij_mj[k]);

			accelerationsGPU_at_i =accelerationsGPU_at_i + diff[k] * dij_mj[k];
		}
	}

	// additional values are put in a loop
	float3 diff;
	float dij;
	float dij_mj;
	int number_loop=(max_j-min_j)/number_unrolling;
	int next_value=min_j+number_loop*number_unrolling;
	for(int j =next_value; j<max_j; j++){
		compute_difference(position_at_i, positionsGPU[j], diff, dij);
		compute_forces(massesGPU[j], dij, dij_mj);
		accelerationsGPU_at_i =accelerationsGPU_at_i + diff * dij_mj;
	}

		
}



template<const int number_unrolling, const int number_particles_shared_memory>
__global__ void update_acc(float3 * positionsGPU, float3 * velocitiesGPU,\
	 float3 * accelerationsGPU,float* massesGPU,const int n_particles)
{	

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (i >= n_particles) return;
	//should be useless, except for debug purpose
	accelerationsGPU[i] = make_float3(0.0f, 0.0f, 0.0f);


	// shared memory
	__shared__ float3 positions_shared[number_particles_shared_memory];
	__shared__ float masses_shared[number_particles_shared_memory];
	//note that acceleration is not shared since it is acess once per thread
	// this is also the case for the position at index i

	int number_mem_load=(n_particles+number_particles_shared_memory-1)/number_particles_shared_memory;
	for (int mem_load_id=0; mem_load_id<number_mem_load;mem_load_id++){
		
		// copy the data in the shared memory
		int j=mem_load_id*number_particles_shared_memory;
		
		for (int k=0; k<number_particles_shared_memory; k++){
			if (j+k<n_particles){
				positions_shared[k]=positionsGPU[j+k];
				masses_shared[k]=massesGPU[j+k];
			}
		}


		__syncthreads();
		// update the acceleration
		int min_j= 0;
		int max_j= number_particles_shared_memory;
		
		if (j+max_j>n_particles) max_j=n_particles-j;
		update_acc_at_i_between_j<number_unrolling>(
			positionsGPU[i], accelerationsGPU[i],
			positions_shared,  masses_shared,\
		min_j, max_j);
		__syncthreads();

	}

	// update_acc_at_i_between_j<number_unrolling>(positionsGPU, velocitiesGPU, accelerationsGPU, massesGPU,\
	// i, 0, n_particles);
	
	

}


// __global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
// {
// 	// unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

// }

static inline int divup(int a, int b) {
	// how many blocks of size b should we use to represent a block of size a
	return (a + b - 1)/b;
}



void update_position_cu(float3* positionsGPU,float3* velocitiesGPU, \
	float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	int nthreads = 128;
	int nblocks = divup(n_particles, nthreads);

	update_acc<3,256> <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU,\
	 massesGPU, n_particles);

	

}


#endif // GALAX_MODEL_GPU